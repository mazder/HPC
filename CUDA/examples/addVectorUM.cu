
#include <hip/hip_runtime.h>
#include<iostream>
#include<omp.h>
#include<cassert>

void printV(int *a, int *b, int *c, int N ){
    for(int i=0; i<10; i++){
        printf("%d %d %d\n",a[i],b[i],c[i]);
    }
}

void initVector(int* a, int* b, int N){
    for(int i=0; i<N; i++){
        a[i]=i%2;
        b[i]=i%2;
    }
}

void check(int* a, int* b, int* c, int N){
    for(int i=0; i<N; i++){
        assert(c[i]==a[i]+b[i]);
    }
    std::cout<<"Verification Success"<<std::endl;
}

__global__ void vectorAdd(int *A, int* B, int* C, int N){

    int tid = (blockIdx.x*blockDim.x)+threadIdx.x;

    if (tid<N){
        C[tid]=A[tid]+B[tid];
    }
}

// vector add
void vectorAddUM(){

    // Number of elements 2^16 (65536 elements)
    int N = 1<<16;

    //size_t bytes = N*sizeof(int);

    int *a_u, *b_u, *c_u; // host & device pointers



    // allocate in device
    hipMallocManaged(&a_u, N*sizeof(int));
    hipMallocManaged(&b_u, N*sizeof(int));
    hipMallocManaged(&c_u, N*sizeof(int));

    // initialize a_u and b_u
    initVector(a_u, b_u, N);

    int blockSize = 256; // Dg, number of threads per block
    int numBlocks = (N+blockSize -1)/blockSize; // Db, number of blocks
    vectorAdd<<<numBlocks,blockSize>>>(a_u, b_u, c_u, N);

    hipDeviceSynchronize();

    check(a_u, b_u, c_u, N);

    // deallocate memory
    hipFree(a_u);
    hipFree(b_u);
    hipFree(c_u);
    std::cout<<"Completed Successfully"<<std::endl;
}


int main(int argc, char* argv[])
{

    vectorAddUM();

    return 0;
}
