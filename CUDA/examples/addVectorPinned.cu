
#include <hip/hip_runtime.h>
#include<iostream>
#include<omp.h>
#include<cassert>

void printV(int *a, int *b, int *c, int N ){
    for(int i=0; i<10; i++){
        printf("%d %d %d\n",a[i],b[i],c[i]);
    }
}

void initVector(int* a, int* b, int N){
    for(int i=0; i<N; i++){
        a[i]=i%2;
        b[i]=i%2;
    }
}

void check(int* a, int* b, int* c, int N){
    for(int i=0; i<N; i++){
        assert(c[i]==a[i]+b[i]);
    }
    std::cout<<"Verification Success"<<std::endl;
}


__global__ void vectorAdd(int *A, int* B, int* C, int N){

    int tid = (blockIdx.x*blockDim.x)+threadIdx.x;

    if (tid<N){
        C[tid]=A[tid]+B[tid];
    }
}


// vector add
void vectorAddNative(){

    // Number of elements
    int N = 17; //1<<16;

    //size_t bytes = N*sizeof(int);

    int *a_h, *b_h, *c_h; // host pointers
    int *a_d, *b_d, *c_d; // device pointers

    // allocate in host memory // usually pagable allocation
    /*
    a_h = (int *) malloc(N*sizeof(int));
    b_h = (int *) malloc(N*sizeof(int));
    c_h = (int *) malloc(N*sizeof(int));
    */
    hipHostMalloc(&a_h, N*sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&b_h, N*sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&c_h, N*sizeof(int), hipHostMallocDefault);

    // initialize a_h and b_h
    initVector(a_h, b_h, N);

    // allocate in device
    hipMalloc(&a_d, N*sizeof(int));
    hipMalloc(&b_d, N*sizeof(int));
    hipMalloc(&c_d, N*sizeof(int));

    // Host to Device
    hipMemcpy(a_d, a_h, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, N*sizeof(int), hipMemcpyHostToDevice);


    //<<< Dg, Db, Ns, S >>> blockDim()

    int blockSize = 256; // Dg, number of threads per block
    int numBlocks = (N+blockSize -1)/blockSize; // Db, number of blocks
    vectorAdd<<<numBlocks,blockSize>>>(a_d, b_d, c_d, N);

    hipDeviceSynchronize();

    // Device to Host
    hipMemcpy(c_h, c_d, N*sizeof(int), hipMemcpyDeviceToHost);

    check(a_h, b_h, c_h, N);

    // deallocate host memory
    hipHostFree(a_h);
    hipHostFree(b_h);
    hipHostFree(c_h);

    // deallocate device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

}


int main(int argc, char* argv[])
{
    //hello();

    vectorAddNative();

    return 0;
}

